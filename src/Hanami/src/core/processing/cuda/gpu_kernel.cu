#include "hip/hip_runtime.h"
/**
 * @file        gpu_kernel.cu
 *
 * @author      Tobias Anker <tobias.anker@kitsunemimi.moe>
 *
 * @copyright   Apache License Version 2.0
 *
 *      Copyright 2022 Tobias Anker
 *
 *      Licensed under the Apache License, Version 2.0 (the "License");
 *      you may not use this file except in compliance with the License.
 *      You may obtain a copy of the License at
 *
 *          http://www.apache.org/licenses/LICENSE-2.0
 *
 *      Unless required by applicable law or agreed to in writing, software
 *      distributed under the License is distributed on an "AS IS" BASIS,
 *      WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *      See the License for the specific language governing permissions and
 *      limitations under the License.
 */

#include <iostream>
#include <chrono>
#include <math.h>

#include "../objects.h"
#include "../cluster_io_functions.h"

//==================================================================================================
//==================================================================================================
//==================================================================================================

/**
 * @brief initialize a new synpase
 *
 * @param block source-neuron-block, which is only used to hold the randamo-value
 * @param synapse pointer to the synapse, which should be (re-) initialized
 * @param clusterSettings pointer to the cluster-settings
 * @param remainingW new weight for the synapse
 * @param randomValues pointer to the buffer with all randow-values
 */
__device__ __forceinline__ void
createNewSynapse(NeuronBlock* block,
                 Synapse* synapse,
                 const ClusterSettings* clusterSettings,
                 const float remainingW,
                 const uint32_t* randomValues)
{
    uint32_t randomPos = (block->randomPos + (threadIdx.x * blockIdx.x) + 1)
                               % (NUMBER_OF_RAND_VALUES - 5);
    block->randomPos = randomPos;

    const float randMax = static_cast<float>(RAND_MAX);
    uint32_t signRand = 0;
    const float sigNeg = clusterSettings->signNeg;

    // set activation-border
    synapse->border = remainingW;

    // set target neuron
    randomPos = (randomPos + 1) % NUMBER_OF_RAND_VALUES;
    synapse->targetNeuronId
        = static_cast<uint16_t>(randomValues[randomPos] % block->numberOfNeurons);

    randomPos = (randomPos + 1) % NUMBER_OF_RAND_VALUES;
    synapse->weight = (static_cast<float>(randomValues[randomPos]) / randMax) / 10.0f;

    // update weight with sign
    randomPos = (randomPos + 1) % NUMBER_OF_RAND_VALUES;
    signRand = randomValues[randomPos] % 1000;
    synapse->weight *= static_cast<float>(1.0f - (1000.0f * sigNeg > signRand) * 2);
}

/**
 * @brief process a single synapse-section
 *
 * @param synapseSection current synapse-section to process
 * @param connection pointer to the connection-object, which is related to the section
 * @param targetNeuronBlock neuron-block, which is the target for all synapses in the section
 * @param sourceNeuron pointer to source-neuron, which had triggered the section
 * @param originLocation location of the source-neuron to mark updates
 * @param clusterSettings pointer to cluster-settings
 * @param randomValues pointer to the list with all random-values
 * @param localMem pointer to shared-memory, which should be used by the processing thread
 */
template <bool doTrain>
__device__ __forceinline__ void
synapseProcessingBackward(SynapseSection* synapseSection,
                          SynapseConnection* connection,
                          NeuronBlock* targetNeuronBlock,
                          Neuron* sourceNeuron,
                          const SourceLocationPtr originLocation,
                          ClusterSettings* clusterSettings,
                          const uint* randomValues,
                          float* localMem)
{
    __shared__ float localPotential[64];
    localPotential[threadIdx.x] = sourceNeuron->potential - connection->offset;

    float val = 0.0f;
    uint16_t pos = 0;
    Synapse* synapse = nullptr;

    // iterate over all synapses in the section
    while (pos < SYNAPSES_PER_SYNAPSESECTION && localPotential[threadIdx.x] > 0.01f) {
        synapse = &synapseSection->synapses[pos];

        if constexpr (doTrain) {
            // create new synapse if necesarry and training is active
            if (synapse->targetNeuronId == UNINIT_STATE_16) {
                createNewSynapse(targetNeuronBlock,
                                 synapse,
                                 clusterSettings,
                                 localPotential[threadIdx.x],
                                 randomValues);
            }

            // split synapse, if necessary
            if (synapse->border > 2.0f * localPotential[threadIdx.x]
                    && pos < SYNAPSES_PER_SYNAPSESECTION - 2)
            {
                const float val = synapse->border / 2.0f;
                synapseSection->synapses[pos + 1].border += val;
                synapse->border -= val;
            }
        }

        if (synapse->targetNeuronId != UNINIT_STATE_16) {
            // update target-neuron
            val = synapse->weight;
            if (localPotential[threadIdx.x] < synapse->border) {
                val *= ((1.0f / synapse->border) * localPotential[threadIdx.x]);
            }
            localMem[synapse->targetNeuronId] += val;
        }

        // update loop-counter
        localPotential[threadIdx.x] -= synapse->border;
        ++pos;
    }

    // mark source-neuron for updates, if necessary and training is active
    if constexpr (doTrain) {
        sourceNeuron->isNew = localPotential[threadIdx.x] > 0.01f && synapseSection->hasNext == false;
        sourceNeuron->newOffset = (sourceNeuron->potential - localPotential[threadIdx.x]) + connection->offset;
        synapseSection->hasNext = synapseSection->hasNext || sourceNeuron->isNew;
    }
}

/**
 * @brief processSynapses
 *
 * @param neuronBlocks pointer to neuron-blocks in gpu-memory
 * @param synapseBlocks pointer to synapse-blocks in gpu-memory
 * @param connectionBlocks pointer to connection-blocks in gpu-memory
 * @param clusterSettings pointer to cluster-settingss in gpu-memory
 * @param randomValues pointer to list with random-values in gpu-memory
 * @param neuronBlockPos position-offset within the neuron-block-buffer
 * @param dimY number of connections-blocks in y-direction
 */
template <bool doTrain>
__global__ void
processSynapses(NeuronBlock* neuronBlocks,
                SynapseBlock* synapseBlocks,
                ConnectionBlock* connectionBlocks,
                ClusterSettings* clusterSettings,
                const uint32_t* randomValues,
                const uint32_t neuronBlockPos,
                const uint32_t dimY)
{
    SynapseBlock* synapseBlock = nullptr;
    const uint64_t tid = threadIdx.x;
    const uint64_t neuronBlockId = (blockIdx.x / dimY) + neuronBlockPos;

    // init temp-values, one for each thread and each neuron
    __shared__ float tempVal[64][64];
    for (uint i = 0; i < 64; ++i){
        tempVal[tid][i] = 0.0f;
    }

    // process synapses
    ConnectionBlock* connectionBlock = &connectionBlocks[blockIdx.x];
    SynapseConnection* scon = &connectionBlock->connections[tid];

    if (connectionBlock->targetSynapseBlockPos != UNINIT_STATE_64) {
        synapseBlock =  &synapseBlocks[connectionBlock->targetSynapseBlockPos];

        if (scon->origin.blockId != UNINIT_STATE_32) {
            NeuronBlock* sourceNeuronBlock = &neuronBlocks[scon->origin.blockId];
            Neuron* sourceNeuron = &sourceNeuronBlock->neurons[scon->origin.neuronId];

            if (sourceNeuron->active != 0) {
                SynapseSection* synapseSection = &synapseBlock->sections[tid];
                NeuronBlock* targetNeuronBlock = &neuronBlocks[neuronBlockId];

                synapseProcessingBackward<doTrain>(synapseSection,
                                                   scon,
                                                   targetNeuronBlock,
                                                   sourceNeuron,
                                                   scon->origin,
                                                   clusterSettings,
                                                   randomValues,
                                                   tempVal[tid]);
            }
        }
    }

    __syncthreads();

    // fill temp-values of the synapse-block
    if (connectionBlock->targetSynapseBlockPos != UNINIT_STATE_64) {
        for (uint i = 0; i < 64; ++i) {
            synapseBlock->tempValues[tid] += tempVal[i][tid];
        }
    }
}

/**
 * @brief process neurons
 *
 * @param neuronBlocks pointer to neuron-blocks in gpu-memory
 * @param synapseBlocks pointer to synapse-blocks in gpu-memory
 * @param connectionBlocks pointer to connection-blocks in gpu-memory
 * @param clusterSettings pointer to cluster-settings in gpu-memory
 * @param neuronBlockPos position-offset within the neuron-block-buffer
 * @param dimY number of connections-blocks in y-direction
 * @param isOutputBrick true, if current brick is an output-brick
 */
template <bool doTrain>
__global__ void
processNeurons(NeuronBlock* neuronBlocks,
               SynapseBlock* synapseBlocks,
               ConnectionBlock* connectionBlocks,
               ClusterSettings* clusterSettings,
               const uint32_t neuronBlockPos,
               const uint32_t dimY,
               const bool isOutputBrick)
{
    // init shared memory
    __shared__ float localInputs[64];
    localInputs[threadIdx.x] = 0.0f;

    // init global pointers
    const uint64_t neuronBlockId = blockIdx.x + neuronBlockPos;
    NeuronBlock* targetNeuronBlock = &neuronBlocks[neuronBlockId];
    Neuron* neuron = &targetNeuronBlock->neurons[threadIdx.x];
    ConnectionBlock* connectionBlock = nullptr;
    SynapseBlock* synapseBlock = nullptr;

    // copy input-values of all releaded synpase-blocks into the neurons
    for (int c = blockIdx.x * dimY; c < (blockIdx.x * dimY) + dimY; ++c) {
        connectionBlock = &connectionBlocks[c];
        if (connectionBlock->targetSynapseBlockPos != UNINIT_STATE_64) {
            synapseBlock =  &synapseBlocks[connectionBlock->targetSynapseBlockPos];
            localInputs[threadIdx.x] += synapseBlock->tempValues[threadIdx.x];
            synapseBlock->tempValues[threadIdx.x] = 0.0f;
        }
    }
    neuron->input = localInputs[threadIdx.x];

    // process neuron-content
    if(isOutputBrick == false)
    {
        neuron->potential /= clusterSettings->neuronCooldown;
        neuron->refractionTime = neuron->refractionTime >> 1;

        if (neuron->refractionTime == 0) {
            neuron->potential = clusterSettings->potentialOverflow * neuron->input;
            neuron->refractionTime = clusterSettings->refractionTime;
        }

        neuron->potential -= neuron->border;
        neuron->active = neuron->potential > 0.0f;
        neuron->input = 0.0f;
        neuron->potential = log2(neuron->potential + 1.0f);

        if constexpr (doTrain) {
            neuron->isNew = neuron->active != 0 && neuron->inUse == 0;
            neuron->newOffset = 0.0f;
        }
    }
}

//==================================================================================================
//==================================================================================================
//==================================================================================================

/**
 * @brief process neuron in backpropagation-steup
 *
 * @param neuronBlocks pointer to neuron-blocks in gpu-memory
 * @param tempNeuronBlocks pointer to temp-buffer of neuron-blocks in gpu-memory
 * @param neuronBlockPos position-offset within the neuron-block-buffer
 */
__global__ void
backpropagateNeurons(NeuronBlock* neuronBlocks,
                     TempNeuronBlock* tempNeuronBlocks,
                     const uint32_t neuronBlockPos)
{
    __shared__ float localDelta[64];

    const uint64_t neuronBlockId = blockIdx.x + neuronBlockPos;
    const NeuronBlock* targetNeuronBlock = &neuronBlocks[neuronBlockId];
    const Neuron* targetNeuron = &targetNeuronBlock->neurons[threadIdx.x];
    TempNeuronBlock* targetTempBlock = &tempNeuronBlocks[neuronBlockId];
    TempNeuron* targetTempNeuron = &targetTempBlock->neurons[threadIdx.x];

    if (targetNeuron->active) {
        // aggregate different delta-values
        localDelta[threadIdx.x] = 0.0f;
        localDelta[threadIdx.x] += targetTempNeuron->delta[0];
        localDelta[threadIdx.x] += targetTempNeuron->delta[1];
        localDelta[threadIdx.x] += targetTempNeuron->delta[2];
        localDelta[threadIdx.x] += targetTempNeuron->delta[3];
        localDelta[threadIdx.x] += targetTempNeuron->delta[4];
        localDelta[threadIdx.x] += targetTempNeuron->delta[5];
        localDelta[threadIdx.x] += targetTempNeuron->delta[6];
        localDelta[threadIdx.x] += targetTempNeuron->delta[7];

        // calculate new delta-value for next iteration
        localDelta[threadIdx.x] *= 1.4427f * pow(0.5f, targetNeuron->potential);
        targetTempNeuron->delta[0] = localDelta[threadIdx.x];
    }
}

/**
 * @brief backpropagate a synapse-section
 *
 * @param section current synapse-section
 * @param connection current connection related to the synapse-section
 * @param targetTempBlock temp-value-block of the target neuron-block
 * @param sourceNeuron source-neuron, which triggered the section
 * @param sourceTempNeuron temp-balue block of the source-neuron
 */
__device__ __forceinline__ void
backpropagateSection(SynapseSection* section,
                     SynapseConnection* connection,
                     TempNeuronBlock* targetTempBlock,
                     Neuron* sourceNeuron,
                     TempNeuron* sourceTempNeuron)
{
    __shared__ float localDelta[64];
    __shared__ float localTotalDeltas[64];
    __shared__ float localPotential[64];

    // init values
    localPotential[threadIdx.x] = sourceNeuron->potential - connection->offset;
    Synapse* synapse = nullptr;
    TempNeuron* targetTempNeuron = nullptr;
    constexpr float trainValue = 0.05f;
    localTotalDeltas[threadIdx.x] = 0.0f;
    float valid = 0.0f;

    // iterate over all synapses in the section
    for (uint16_t pos = 0; pos < SYNAPSES_PER_SYNAPSESECTION; pos++) {
        synapse = &section->synapses[pos];

        if (synapse->targetNeuronId != UNINIT_STATE_16) {
            targetTempNeuron = &targetTempBlock->neurons[synapse->targetNeuronId];

            // calculate new delta
            localDelta[threadIdx.x] = targetTempNeuron->delta[0] * synapse->weight;
            if (localPotential[threadIdx.x] < synapse->border) {
                localDelta[threadIdx.x] *= (1.0f / synapse->border) * localPotential[threadIdx.x];
            }

            // update values
            valid = (float)(localPotential[threadIdx.x] > 0.01f);
            synapse->weight -= trainValue * targetTempNeuron->delta[0] * valid;
            localTotalDeltas[threadIdx.x] += localDelta[threadIdx.x] * valid;
            localPotential[threadIdx.x] -= synapse->border;
        }
    }

    sourceTempNeuron->delta[connection->origin.posInNeuron] = localTotalDeltas[threadIdx.x];
}

/**
 * @brief backpropagate connections
 *
 * @param neuronBlocks pointer to neuron-blocks in gpu-memory
 * @param tempNeuronBlocks pointer to temp-values of the neuron-blocks in gpu-memory
 * @param synapseBlocks pointer to synapse-blocks in gpu-memory
 * @param connectionBlocks pointer to connection-blocks in gpu-memory
 * @param neuronBlockPos position-offset within the neuron-block-buffer
 * @param dimY number of connections-blocks in y-direction
 */
__global__ void
backpropagateConnections(NeuronBlock* neuronBlocks,
                         TempNeuronBlock* tempNeuronBlocks,
                         SynapseBlock* synapseBlocks,
                         ConnectionBlock* connectionBlocks,
                         const uint32_t neuronBlockPos,
                         const uint32_t dimY)
{
    ConnectionBlock* connectionBlock = &connectionBlocks[blockIdx.x];
    SynapseConnection* scon = &connectionBlock->connections[threadIdx.x];

    if (scon->origin.blockId != UNINIT_STATE_32) {
        SynapseSection* synapseSection = &synapseBlocks[connectionBlock->targetSynapseBlockPos].sections[threadIdx.x];

        NeuronBlock* sourceNeuronBlock = &neuronBlocks[scon->origin.blockId];
        TempNeuronBlock* sourceTempBlock = &tempNeuronBlocks[scon->origin.blockId];
        Neuron* sourceNeuron = &sourceNeuronBlock->neurons[scon->origin.neuronId];
        TempNeuron* sourceTempNeuron = &sourceTempBlock->neurons[scon->origin.neuronId];

        const uint64_t neuronBlockId = (blockIdx.x / dimY)  + neuronBlockPos;
        TempNeuronBlock* targetTempBlock = &tempNeuronBlocks[neuronBlockId];

        backpropagateSection(synapseSection, scon, targetTempBlock, sourceNeuron, sourceTempNeuron);
    }
}

//==================================================================================================
//==================================================================================================
//==================================================================================================

/**
 * @brief initial copy of data from the host to the gpu
 *
 * @param gpuPointer pointer to the handle-object, which will store the pointer for the gpu-buffer
 * @param clusterSettings pointer to cluster-settings on host
 * @param neuronBlocks pointer to neuron-blocks on host
 * @param tempNeuronBlocks pointer to temp-values of the neuron-blocks on host
 * @param numberOfNeuronBlocks number of neuron-blocks to copy
 * @param synapseBlocks pointer to synapse-blocks on host
 * @param numberOfSynapseBlocks number of synapse-blocks to copy
 * @param bricks pointer to bricks to initialize their connection-blocks, if exist
 * @param numberOfBricks number of bricks in the cluster to init the connection-block-buffer
 * @param randomValues pointer to neuron-blocks on host
 */
extern "C"
void
copyToDevice_CUDA(CudaPointerHandle* gpuPointer,
                  ClusterSettings* clusterSettings,
                  NeuronBlock* neuronBlocks,
                  TempNeuronBlock* tempNeuronBlocks,
                  const uint32_t numberOfNeuronBlocks,
                  SynapseBlock* synapseBlocks,
                  const uint32_t numberOfSynapseBlocks,
                  Brick* bricks,
                  const uint32_t numberOfBricks,
                  uint32_t* randomValues)
{
    // allocate memory on gpu
    hipMalloc(&gpuPointer->clusterSettings, 1                     * sizeof(ClusterSettings));
    hipMalloc(&gpuPointer->neuronBlocks,    numberOfNeuronBlocks  * sizeof(NeuronBlock));
    hipMalloc(&gpuPointer->tempNeuronBlock, numberOfNeuronBlocks  * sizeof(TempNeuronBlock));
    hipMalloc(&gpuPointer->synapseBlocks,   numberOfSynapseBlocks * sizeof(SynapseBlock));
    hipMalloc(&gpuPointer->randomValues,    NUMBER_OF_RAND_VALUES * sizeof(uint32_t));

    // copy data from host into the allocated memory
    hipMemcpy(gpuPointer->clusterSettings, clusterSettings,  1                     * sizeof(ClusterSettings), hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->neuronBlocks,    neuronBlocks,     numberOfNeuronBlocks  * sizeof(NeuronBlock),     hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->tempNeuronBlock, tempNeuronBlocks, numberOfNeuronBlocks  * sizeof(TempNeuronBlock), hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->synapseBlocks,   synapseBlocks,    numberOfSynapseBlocks * sizeof(SynapseBlock),    hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->randomValues,    randomValues,     NUMBER_OF_RAND_VALUES * sizeof(uint32_t),        hipMemcpyHostToDevice);

    // initialize connection-blocks all all bricks
    gpuPointer->connectionBlocks.resize(numberOfBricks);
    for (uint32_t brickId = 0; brickId < numberOfBricks; ++brickId) {
        gpuPointer->connectionBlocks[brickId] = nullptr;

        Brick* brick = &bricks[brickId];
        if(brick->connectionBlocks.size() > 0) {
            hipMalloc(&gpuPointer->connectionBlocks[brickId],
                       brick->connectionBlocks.size() * sizeof(ConnectionBlock));
            hipMemcpy(gpuPointer->connectionBlocks[brickId],
                       &brick->connectionBlocks[0],
                       brick->connectionBlocks.size() * sizeof(ConnectionBlock),
                       hipMemcpyHostToDevice);
        }
    }
}

/**
 * @brief removed all data from the gpu, which are linked in the handle-object
 *
 * @param gpuPointer handle with all pointer to free
 */
extern "C"
void
removeFromDevice_CUDA(CudaPointerHandle* gpuPointer)
{
    for (uint32_t c = 0; c < gpuPointer->connectionBlocks.size(); ++c)
    {
        // free old connection-block-memory on gpu, if exist
        if (gpuPointer->connectionBlocks[c] != nullptr)
        {
            hipFree(gpuPointer->connectionBlocks[c]);
            gpuPointer->connectionBlocks[c] = nullptr;
        }
    }

    hipFree(gpuPointer->clusterSettings);
    hipFree(gpuPointer->neuronBlocks);
    hipFree(gpuPointer->tempNeuronBlock);
    hipFree(gpuPointer->synapseBlocks);
    hipFree(gpuPointer->randomValues);
}

/**
 * @brief copy all data from the gpu back to the host
 *
 * @param gpuPointer handle with all gpu-pointer of the cluster
 * @param neuronBlocks pointer to neuron-blocks on host
 * @param numberOfNeuronBlocks number of neuron-blocks to copy
 * @param synapseBlocks pointer to synpase-blocks on host
 * @param numberOfSynapseBlocks number of synpase-blocks to copy
 */
extern "C"
void
copyFromGpu_CUDA(CudaPointerHandle* gpuPointer,
                 NeuronBlock* neuronBlocks,
                 const uint32_t numberOfNeuronBlocks,
                 SynapseBlock* synapseBlocks,
                 const uint32_t numberOfSynapseBlocks)
{
    hipMemcpy(neuronBlocks,
               gpuPointer->neuronBlocks,
               numberOfNeuronBlocks * sizeof(NeuronBlock),
               hipMemcpyDeviceToHost);
    hipMemcpy(synapseBlocks,
               gpuPointer->synapseBlocks,
               numberOfSynapseBlocks * sizeof(SynapseBlock),
               hipMemcpyDeviceToHost);
}

/**
 * @brief in case the cluster was resized, these changes have to be pushed to the gpu
 *
 * @param gpuPointer handle with all gpu-pointer of the cluster
 * @param neuronBlocks pointer to local buffer with neuron-blocks to update
 * @param numberOfNeuronBlocks number of neuron-blocks to update
 * @param bricks pointer to local bricks to access and update their connection-blocks
 * @param numberOfBricks number of bricks to update
 */
extern "C"
void
update_CUDA(CudaPointerHandle* gpuPointer,
            NeuronBlock* neuronBlocks,
            const uint32_t numberOfNeuronBlocks,
            Brick* bricks,
            const uint32_t numberOfBricks)
{
    for (uint32_t brickId = 0; brickId < numberOfBricks; ++brickId)
    {
        Brick* brick = &bricks[brickId];

        if (brick->wasResized) {
            // free old connection-block-memory on gpu, if exist
            if (gpuPointer->connectionBlocks[brickId] != nullptr)
            {
                hipFree(gpuPointer->connectionBlocks[brickId]);
                gpuPointer->connectionBlocks[brickId] = nullptr;
            }

            // allocate to resized memory for the connectionblocks on gpu
            hipMalloc(&gpuPointer->connectionBlocks[brickId],
                       brick->connectionBlocks.size() * sizeof(ConnectionBlock));
        }

        hipMemcpy(gpuPointer->connectionBlocks[brickId],
                   &brick->connectionBlocks[0],
                   brick->connectionBlocks.size() * sizeof(ConnectionBlock),
                   hipMemcpyHostToDevice);

        brick->wasResized = false;
    }
}

/**
 * @brief process all normal- and output-bricks and train them, if wanted.
 *
 * @param gpuPointer handle with all gpu-pointer of the cluster
 * @param bricks pointer to local bricks
 * @param numberOfBricks number of bricks
 * @param neuronBlocks pointer to local neuron-block
 * @param numberOfNeuronBlocks number of neuron-blokcs
 * @param doTrain true to run a taining-process
 */
extern "C"
void
processing_CUDA(CudaPointerHandle* gpuPointer,
                Brick* bricks,
                const uint32_t numberOfBricks,
                NeuronBlock* neuronBlocks,
                const uint32_t numberOfNeuronBlocks,
                const bool doTrain)
{
    // copy necessary data from host to gpu
    hipMemcpy(gpuPointer->neuronBlocks,
               neuronBlocks,
               numberOfNeuronBlocks * sizeof(NeuronBlock),
               hipMemcpyHostToDevice);

    // process bricks on gpu
    for (uint32_t brickId = 0; brickId < numberOfBricks; ++brickId)
    {
        Brick* brick = &bricks[brickId];
        if (brick->isInputBrick) {
            continue;
        }

        if (doTrain)
        {
            processSynapses<true><<<brick->dimX * brick->dimY, 64>>>(
                gpuPointer->neuronBlocks,
                gpuPointer->synapseBlocks,
                gpuPointer->connectionBlocks[brickId],
                gpuPointer->clusterSettings,
                gpuPointer->randomValues,
                brick->neuronBlockPos,
                brick->dimY);

            processNeurons<true><<<brick->dimX, 64>>>(
                gpuPointer->neuronBlocks,
                gpuPointer->synapseBlocks,
                gpuPointer->connectionBlocks[brickId],
                gpuPointer->clusterSettings,
                brick->neuronBlockPos,
                brick->dimY,
                brick->isOutputBrick);
        }
        else
        {
            processSynapses<false><<<brick->dimX * brick->dimY, 64>>>(
                gpuPointer->neuronBlocks,
                gpuPointer->synapseBlocks,
                gpuPointer->connectionBlocks[brickId],
                gpuPointer->clusterSettings,
                gpuPointer->randomValues,
                brick->neuronBlockPos,
                brick->dimY);

            processNeurons<false><<<brick->dimX, 64>>>(
                gpuPointer->neuronBlocks,
                gpuPointer->synapseBlocks,
                gpuPointer->connectionBlocks[brickId],
                gpuPointer->clusterSettings,
                brick->neuronBlockPos,
                brick->dimY,
                brick->isOutputBrick);
        }
    }

    // copy resulting data back to host
    hipMemcpy(neuronBlocks,
               gpuPointer->neuronBlocks,
               numberOfNeuronBlocks * sizeof(NeuronBlock),
               hipMemcpyDeviceToHost);
}

/**
 * @brief run backpropagaion on all normal- and output-brikcs to update the weights
 *        of the synapses.
 *
 * @param gpuPointer handle with all gpu-pointer of the cluster
 * @param bricks pointer to local bricks
 * @param numberOfBricks number of bricks
 * @param neuronBlocks pointer to local neuron-blocks
 * @param tempNeuronBlocks pointer to local temp-values of the neuron-blocks
 * @param numberOfNeuronBlocks number of neuron-blocks
 */
extern "C"
void
backpropagation_CUDA(CudaPointerHandle* gpuPointer,
                     Brick* bricks,
                     const uint32_t numberOfBricks,
                     NeuronBlock* neuronBlocks,
                     TempNeuronBlock* tempNeuronBlocks,
                     const uint32_t numberOfNeuronBlocks)
{
    // copy necessary data from host to gpu
    hipMemcpy(gpuPointer->neuronBlocks,
               neuronBlocks,
               numberOfNeuronBlocks * sizeof(NeuronBlock),
               hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->tempNeuronBlock,
               tempNeuronBlocks,
               numberOfNeuronBlocks * sizeof(TempNeuronBlock),
               hipMemcpyHostToDevice);

    // process all bricks on gpu
    for (int32_t brickId = numberOfBricks - 1; brickId >= 0; --brickId)
    {
        Brick* brick = &bricks[brickId];
        if (brick->isInputBrick) {
            continue;
        }

        backpropagateNeurons<<<brick->dimX, 64>>>(
                gpuPointer->neuronBlocks,
                gpuPointer->tempNeuronBlock,
                brick->neuronBlockPos);

        backpropagateConnections<<<brick->dimX * brick->dimY, 64>>>(
                gpuPointer->neuronBlocks,
                gpuPointer->tempNeuronBlock,
                gpuPointer->synapseBlocks,
                gpuPointer->connectionBlocks[brickId],
                brick->neuronBlockPos,
                brick->dimY);
    }

    // copy neurons back to host
    hipMemcpy(neuronBlocks,
               gpuPointer->neuronBlocks,
               numberOfNeuronBlocks * sizeof(NeuronBlock),
               hipMemcpyDeviceToHost);
}
