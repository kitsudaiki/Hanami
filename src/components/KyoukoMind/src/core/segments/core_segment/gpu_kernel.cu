#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "objects.h"

//==================================================================================================
//==================================================================================================
//==================================================================================================

__device__ __forceinline__ int
getBlockId()
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    return index / blockDim.x;
}

/**
 * @brief initialize a new specific synapse
 */
__device__ __forceinline__ void
createNewSynapse(SynapseConnection* connection,
                 Synapse* synapse,
                 const NeuronSection* targetNeuronSection,
                 const SegmentSettings* segmentSettings,
                 const float outH,
                 const uint* randomValues)
{
    const float maxWeight = outH / (float)(segmentSettings->synapseSegmentation);

    // set activation-border
    connection->randomPos = (connection->randomPos + 1) % NUMBER_OF_RAND_VALUES;
    synapse->border = maxWeight * ((float)(randomValues[connection->randomPos]) / (float)(RAND_MAX));

    // set target neuron
    connection->randomPos = (connection->randomPos + 1) % NUMBER_OF_RAND_VALUES;
    synapse->targetNeuronId = (ushort)(randomValues[connection->randomPos]
                              % targetNeuronSection->numberOfNeurons);


    connection->randomPos = (connection->randomPos + 1) % NUMBER_OF_RAND_VALUES;
    synapse->weight = ((float)(randomValues[connection->randomPos]) / (float)(RAND_MAX)) / 10.0f;

    // update weight with sign
    connection->randomPos = (connection->randomPos + 1) % NUMBER_OF_RAND_VALUES;
    const uint signRand = randomValues[connection->randomPos] % 1000;
    synapse->weight *= (float)(1.0f - (1000.0f * segmentSettings->signNeg > signRand) * 2);

    synapse->activeCounter = 1;
}

//==================================================================================================

/**
 * @brief process synapse-section
 */
__device__ __forceinline__ void
synapseProcessingBackward(SynapseSection* section,
                          SynapseConnection* connection,
                          NeuronSection* targetNeuronSection,
                          NeuronSection* neuronSections,
                          UpdatePosSection* updatePosSections,
                          SegmentSettings* segmentSettings,
                          const uint* randomValues,
                          float* localMem)
{
    NeuronSection* sourceNeuronSection = &neuronSections[connection->sourceNeuronSectionId];
    Neuron* sourceNeuron = &sourceNeuronSection->neurons[connection->sourceNeuronId];
    const float sourcePotential = sourceNeuron->potential;

    float counter = connection->offset;
    uint pos = 0;

    // iterate over all synapses in the section
    while(pos < SYNAPSES_PER_SYNAPSESECTION
          && sourcePotential > counter)
    {
        Synapse* synapse = &section->synapses[pos];

        // create new synapse if necesarry and learning is active
        if(synapse->targetNeuronId == UNINIT_STATE_16)
        {
            createNewSynapse(connection,
                             synapse,
                             targetNeuronSection,
                             segmentSettings,
                             sourcePotential,
                             randomValues);
        }

        // update target-neuron
        Neuron* targetNeuron = &targetNeuronSection->neurons[synapse->targetNeuronId];
        //targetNeuron->input += synapse->weight;
        localMem[synapse->targetNeuronId] += synapse->weight;

        // update active-counter
        const uint8_t active = (synapse->weight > 0) == (targetNeuron->potential > targetNeuron->border);
        synapse->activeCounter += active * (uint8_t)(synapse->activeCounter < 126);

        // update loop-counter
        counter += synapse->border;
        pos++;
    }

    UpdatePosSection* updateSection = &updatePosSections[connection->sourceNeuronSectionId];
    UpdatePos* updatePos = &updateSection->positions[connection->sourceNeuronId];
    updatePos->type = sourcePotential - counter > 0.01f && connection->forwardNextId == UNINIT_STATE_32;
    updatePos->offset = counter + connection->offset;
}

//==================================================================================================

__device__ __forceinline__ void
prcessNeuronConnection(const uint neuronSectionId,
                       NeuronSection* targetNeuronSection,
                       NeuronConnection* neuronConnections,
                       NeuronSection* neuronSections,
                       SynapseConnection* synapseConnections,
                       SynapseSection* synapseSections,
                       UpdatePosSection* updatePosSections,
                       SegmentSettings* segmentSettings,
                       const uint* randomValues,
                       float* localMem,
                       Neuron* localNeurons,
                       SynapseConnection* localConnections)
{
    // reset weight of neurons
    for(uint neuronId = threadIdx.x;
        neuronId < targetNeuronSection->numberOfNeurons;
        neuronId += blockDim.x)
    {
        targetNeuronSection->neurons[neuronId].input = 0.0f;
    }

    for(uint sectionPos = threadIdx.x;
        sectionPos < NEURON_CONNECTIONS;
        sectionPos += blockDim.x)
    {
        // process synapse-sections
        const uint offset = threadIdx.x * NEURONS_PER_NEURONSECTION;
        const uint sectionId = neuronConnections[neuronSectionId].backwardIds[sectionPos];
        if(sectionId != UNINIT_STATE_32)
        {
            localConnections[threadIdx.x] = synapseConnections[sectionId];
            synapseProcessingBackward(&synapseSections[sectionId],
                                      &localConnections[threadIdx.x],
                                      targetNeuronSection,
                                      neuronSections,
                                      updatePosSections,
                                      segmentSettings,
                                      randomValues,
                                      &localMem[offset]);
        }

        __syncthreads();

        // apply values of the local-memory to the neurons
        for(uint neuronId = threadIdx.x;
            neuronId < targetNeuronSection->numberOfNeurons;
            neuronId += blockDim.x)
        {
            localNeurons[threadIdx.x] = targetNeuronSection->neurons[neuronId];
            Neuron* neuron = &localNeurons[threadIdx.x];
            for(uint i = neuronId;
                i < NEURONS_PER_NEURONSECTION * blockDim.x;
                i += NEURONS_PER_NEURONSECTION)
            {
                neuron->input += localMem[i];
                localMem[i] = 0.0f;
            }
            targetNeuronSection->neurons[neuronId] = localNeurons[threadIdx.x];
        }
    }

    __syncthreads();
}

//==================================================================================================

__device__ __forceinline__ void
resetLocalMemory(float* localMem, const int localSize)
{
    // reset local memory
    for(uint i = threadIdx.x;
        i < localSize;
        i += blockDim.x)
    {
        localMem[i] = 0.0f;
    }
}

//==================================================================================================

/**
 * @brief process all neurons within a segment
 */
__global__ void
prcessCoreSegmentKernel(BrickHeader* bricks,
                        NeuronConnection* neuronConnections,
                        NeuronSection* neuronSections,
                        SynapseConnection* synapseConnections,
                        SynapseSection* synapseSections,
                        UpdatePosSection* updatePosSections,
                        SegmentSettings* segmentSettings,
                        float* inputTransfers,
                        float* outputTransfers,
                        const uint* randomValues,
                        const ulong brickId)
{
    __shared__ uint8_t localMem[4096 * 4 + 64 * sizeof(Neuron) + 64 * sizeof(SynapseConnection)];
    float* localValues = (float*)&localMem[0];
    Neuron* localNeurons = (Neuron*)&localMem[16384];
    SynapseConnection* localConnections = (SynapseConnection*)&localMem[18432];

    resetLocalMemory(localValues, blockDim.x * NEURONS_PER_NEURONSECTION);

    BrickHeader* brick = &bricks[brickId];
    if(brick->isInputBrick == false
            && brick->isOutputBrick == false)
    {
        for(uint neuronSectionId = brick->neuronSectionPos + getBlockId();
            neuronSectionId < brick->numberOfNeuronSections + brick->neuronSectionPos;
            neuronSectionId += gridDim.x)
        {
            NeuronSection* neuronSection = &neuronSections[neuronSectionId];

            prcessNeuronConnection(neuronSectionId,
                                   neuronSection,
                                   neuronConnections,
                                   neuronSections,
                                   synapseConnections,
                                   synapseSections,
                                   updatePosSections,
                                   segmentSettings,
                                   randomValues,
                                   localValues,
                                   localNeurons,
                                   localConnections);

            for(uint neuronId = threadIdx.x;
                neuronId < neuronSection->numberOfNeurons;
                neuronId += blockDim.x)
            {
                localNeurons[threadIdx.x] = neuronSection->neurons[neuronId];
                Neuron* neuron = &localNeurons[threadIdx.x];

                neuron->potential /= segmentSettings->neuronCooldown;
                neuron->refractionTime = neuron->refractionTime >> 1;

                if(neuron->refractionTime == 0)
                {
                    neuron->potential = segmentSettings->potentialOverflow * neuron->input;
                    neuron->refractionTime = segmentSettings->refractionTime;
                }

                // update neuron
                neuron->potential -= neuron->border;
                neuron->active = neuron->potential > 0.0f;
                neuron->input = 0.0f;
                neuron->potential = log2(neuron->potential + 1.0f);

                // handle active-state
                const bool needUpdate = neuron->active != 0 && neuron->targetSectionId == UNINIT_STATE_32;
                UpdatePos* updatePos = &updatePosSections[neuronSectionId].positions[neuronId];
                updatePos->type = needUpdate;
                updatePos->offset = 0.0f;

                neuronSection->neurons[neuronId] = localNeurons[threadIdx.x];
            }
        }
    }
}

//==================================================================================================

__global__ void
prcessOutputKernel(BrickHeader* bricks,
                   NeuronConnection* neuronConnections,
                   NeuronSection* neuronSections,
                   SynapseConnection* synapseConnections,
                   SynapseSection* synapseSections,
                   UpdatePosSection* updatePosSections,
                   SegmentSettings* segmentSettings,
                   float* outputTransfers,
                   const uint* randomValues)
{
    __shared__ uint8_t localMem[4096 * 4 + 64 * sizeof(Neuron) + 64 * sizeof(SynapseConnection)];
    float* localValues = (float*)&localMem[0];
    Neuron* localNeurons = (Neuron*)&localMem[16384];
    SynapseConnection* localConnections = (SynapseConnection*)&localMem[18432];

    resetLocalMemory(localValues, blockDim.x * NEURONS_PER_NEURONSECTION);

    NeuronSection* neuronSection = &neuronSections[getBlockId()];
    BrickHeader* brick = &bricks[neuronSection->brickId];
    if(brick->isOutputBrick)
    {
        prcessNeuronConnection(getBlockId(),
                               neuronSection,
                               neuronConnections,
                               neuronSections,
                               synapseConnections,
                               synapseSections,
                               updatePosSections,
                               segmentSettings,
                               randomValues,
                               localValues,
                               localNeurons,
                               localConnections);


        for(uint neuronId = threadIdx.x;
            neuronId < neuronSection->numberOfNeurons;
            neuronId += blockDim.x)
        {
            localNeurons[threadIdx.x] = neuronSection->neurons[neuronId];
            Neuron* neuron = &localNeurons[threadIdx.x];

            neuron->potential = segmentSettings->potentialOverflow * neuron->input;
            outputTransfers[neuron->targetBorderId] = neuron->potential;
            neuron->input = 0.0f;

            neuronSection->neurons[neuronId] = localNeurons[threadIdx.x];
        }
    }
}

//==================================================================================================

__global__ void
prcessInputKernel(BrickHeader* bricks,
                  NeuronSection* neuronSections,
                  UpdatePosSection* updatePosSections,
                  float* inputTransfers)
{
    __shared__ Neuron localNeurons[64];

    NeuronSection* neuronSection = &neuronSections[getBlockId()];
    BrickHeader* brick = &bricks[neuronSection->brickId];

    if(brick->isInputBrick
            && threadIdx.x < neuronSection->numberOfNeurons)
    {
        localNeurons[threadIdx.x] = neuronSection->neurons[threadIdx.x];
        Neuron* neuron = &localNeurons[threadIdx.x];

        neuron->potential = inputTransfers[neuron->targetBorderId];
        neuron->active = neuron->potential > 0.0f;
        neuronSection->neurons[threadIdx.x] = localNeurons[threadIdx.x];

        // handle active-state
        const bool needUpdate = neuron->active != 0 && neuron->targetSectionId == UNINIT_STATE_32;
        UpdatePos* updatePos = &updatePosSections[getBlockId()].positions[threadIdx.x];
        updatePos->type = needUpdate;
        updatePos->offset = 0.0f;
    }
}

//==================================================================================================
//==================================================================================================
//==================================================================================================

/**
 * @brief run backpropagation for a single synapse-section
 */
__device__ __forceinline__ uint
backpropagateSection(SynapseSection* section,
                     SynapseConnection* connection,
                     Neuron* sourceNeuron,
                     const float outH,
                     const BrickHeader* brick,
                     NeuronSection* neuronSections,
                     SynapseConnection* synapseConnections,
                     SynapseSection* synapseSections)
{
    NeuronSection* targetNeuronSection = &neuronSections[connection->targetNeuronSectionId];
    float learnValue = 0.2f;
    float counter = connection->offset;

    // iterate over all synapses in the section
    for(uint32_t pos = 0; pos < SYNAPSES_PER_SYNAPSESECTION; pos++)
    {
        // break look, if no more synapses to process
        Synapse* synapse = &section->synapses[pos];

        if(outH > counter)
        {
            // update weight
            learnValue = (float)(126 - synapse->activeCounter) * 0.0002f;
            learnValue += 0.05f;
            Neuron* targetNeuron = &targetNeuronSection->neurons[synapse->targetNeuronId];
            sourceNeuron->delta += targetNeuron->delta * synapse->weight;

            synapse->weight -= learnValue * targetNeuron->delta;
        }

        counter += synapse->border;
    }

    return connection->forwardNextId;
}

//==================================================================================================

/**
 * @brief correct weight of synapses within a segment
 */
__global__ void
reweightCoreSegmentKernel(BrickHeader* bricks,
                          NeuronSection* neuronSections,
                          SynapseConnection* synapseConnections,
                          SynapseSection* synapseSections,
                          SegmentSettings* segmentSettings,
                          float* inputTransfers,
                          float* outputTransfers,
                          const ulong brickId)
{
    __shared__ uint8_t localMem[4096 * 4 + 64 * sizeof(Neuron) + 64 * sizeof(SynapseConnection)];
    float* localValues = (float*)&localMem[0];
    Neuron* localNeurons = (Neuron*)&localMem[16384];
    SynapseConnection* localSections = (SynapseConnection*)&localMem[18432];

    BrickHeader* brick = &bricks[brickId];

    for(uint neuronSectionId = brick->neuronSectionPos + getBlockId();
        neuronSectionId < brick->numberOfNeuronSections + brick->neuronSectionPos;
        neuronSectionId += gridDim.x)
    {
        NeuronSection* neuronSection = &neuronSections[neuronSectionId];
        for(uint neuronId = threadIdx.x;
            neuronId < neuronSection->numberOfNeurons;
            neuronId += blockDim.x)
        {
            localNeurons[threadIdx.x] = neuronSection->neurons[neuronId];
            Neuron* sourceNeuron = &localNeurons[threadIdx.x];
            if(sourceNeuron->targetSectionId != UNINIT_STATE_32)
            {
                sourceNeuron->delta = 0.0f;
                if(sourceNeuron->active)
                {
                    uint nextId = sourceNeuron->targetSectionId;
                    while(nextId != UNINIT_STATE_32)
                    {
                        nextId = backpropagateSection(&synapseSections[nextId],
                                                      &synapseConnections[nextId],
                                                      sourceNeuron,
                                                      sourceNeuron->potential,
                                                      brick,
                                                      neuronSections,
                                                      synapseConnections,
                                                      synapseSections);
                    }

                    sourceNeuron->delta *= 1.4427f * pow(0.5f, sourceNeuron->potential);
                }

                if(brick->isInputBrick) {
                    outputTransfers[sourceNeuron->targetBorderId] = sourceNeuron->delta;
                }
            }

           neuronSection->neurons[neuronId] = localNeurons[threadIdx.x];
        }
    }
}

//==================================================================================================

__global__ void
reweightOutputKernel(BrickHeader* bricks,
                     NeuronSection* neuronSections,
                     float* inputTransfers)
{
    __shared__ Neuron localNeurons[64];

    NeuronSection* neuronSection = &neuronSections[getBlockId()];
    BrickHeader* brick = &bricks[neuronSection->brickId];
    if(brick->isOutputBrick
            && threadIdx.x < neuronSection->numberOfNeurons)
    {
        localNeurons[threadIdx.x] = neuronSection->neurons[threadIdx.x];
        localNeurons[threadIdx.x].delta = inputTransfers[localNeurons[threadIdx.x].targetBorderId];
        inputTransfers[localNeurons[threadIdx.x].targetBorderId] = 0.0f;
        neuronSection->neurons[threadIdx.x] = localNeurons[threadIdx.x];
    }
}

struct PointerHandler
{
    BrickHeader* bricks = nullptr;
    uint32_t* brickOrder = nullptr;
    NeuronSection* neuronSections = nullptr;
    SynapseSection* synapseSections = nullptr;
    SegmentSettings* segmentSettings = nullptr;
    float* inputTransfers = nullptr;
    float* outputTransfers = nullptr;
    UpdatePosSection* updatePosSections = nullptr;
    uint32_t* randomValues = nullptr;
    NeuronConnection* neuronConnections = nullptr;
    SynapseConnection* synapseConnections = nullptr;
};

extern "C"
void
copyToDevice_CUDA(PointerHandler* gpuPointer,
                  SegmentSizes* segmentHeader,
                  SegmentSettings* segmentSettings,
                  BrickHeader* brickHeaders,
                  uint32_t* brickOrder,
                  NeuronSection* neuronSections,
                  SynapseSection* synapseSections,
                  UpdatePosSection* updatePosSections,
                  SynapseConnection* synapseConnections,
                  NeuronConnection* neuronConnections,
                  float* inputTransfers,
                  float* outputTransfers,
                  uint32_t* randomValues)
{
    hipMalloc(&gpuPointer->bricks,             segmentHeader->numberOfBricks             * sizeof(BrickHeader));
    hipMalloc(&gpuPointer->brickOrder,         segmentHeader->numberOfBricks             * sizeof(uint32_t));
    hipMalloc(&gpuPointer->neuronSections,     segmentHeader->numberOfNeuronSections     * sizeof(NeuronSection));
    hipMalloc(&gpuPointer->synapseSections,    segmentHeader->numberOfSynapseSections    * sizeof(SynapseSection));
    hipMalloc(&gpuPointer->segmentSettings,    1                                         * sizeof(SegmentSettings));
    hipMalloc(&gpuPointer->inputTransfers,     segmentHeader->numberOfInputTransfers     * sizeof(float));
    hipMalloc(&gpuPointer->outputTransfers,    segmentHeader->numberOfOutputTransfers    * sizeof(float));
    hipMalloc(&gpuPointer->updatePosSections,  segmentHeader->numberOfUpdatePosSections  * sizeof(UpdatePosSection));
    hipMalloc(&gpuPointer->randomValues,       NUMBER_OF_RAND_VALUES                     * sizeof(uint32_t));
    hipMalloc(&gpuPointer->neuronConnections,  segmentHeader->numberOfNeuronSections     * sizeof(NeuronConnection));
    hipMalloc(&gpuPointer->synapseConnections, segmentHeader->numberOfSynapseSections    * sizeof(SynapseConnection));

    hipMemcpy(gpuPointer->bricks,             brickHeaders,       segmentHeader->numberOfBricks            * sizeof(BrickHeader),       hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->brickOrder,         brickOrder,         segmentHeader->numberOfBricks            * sizeof(uint32_t),          hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->neuronSections,     neuronSections,     segmentHeader->numberOfNeuronSections    * sizeof(NeuronSection),     hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->synapseSections,    synapseSections,    segmentHeader->numberOfSynapseSections   * sizeof(SynapseSection),    hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->segmentSettings,    segmentSettings,    1                                        * sizeof(SegmentSettings),   hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->inputTransfers,     inputTransfers,     segmentHeader->numberOfInputTransfers    * sizeof(float),             hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->outputTransfers,    outputTransfers,    segmentHeader->numberOfOutputTransfers   * sizeof(float),             hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->updatePosSections,  updatePosSections,  segmentHeader->numberOfUpdatePosSections * sizeof(UpdatePosSection),  hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->randomValues,       randomValues,       NUMBER_OF_RAND_VALUES                    * sizeof(uint32_t),          hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->neuronConnections,  neuronConnections,  segmentHeader->numberOfNeuronSections    * sizeof(NeuronConnection),  hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->synapseConnections, synapseConnections, segmentHeader->numberOfSynapseSections   * sizeof(SynapseConnection), hipMemcpyHostToDevice);
}


extern "C"
void
processing_CUDA(PointerHandler* gpuPointer,
                SegmentSizes* segmentHeader,
                uint32_t* brickOrder,
                BrickHeader* bricks,
                float* inputTransfers,
                float* outputTransfers,
                const uint32_t numberOfNeuronSections)
{
    hipMemcpy(gpuPointer->inputTransfers,
               inputTransfers,
               segmentHeader->numberOfInputTransfers * sizeof(float),
               hipMemcpyHostToDevice);

    prcessInputKernel<<<numberOfNeuronSections, NEURONS_PER_NEURONSECTION>>>(
        gpuPointer->bricks,
        gpuPointer->neuronSections,
        gpuPointer->updatePosSections,
        gpuPointer->inputTransfers);

    for(uint32_t pos = 0; pos < segmentHeader->numberOfBricks; pos++)
    {
        const uint32_t brickId = brickOrder[pos];
        BrickHeader* brick = &bricks[brickId];
        if(brick->isInputBrick == false
                && brick->isOutputBrick == false)
        {
            prcessCoreSegmentKernel<<<10, 64>>>(
                gpuPointer->bricks,
                gpuPointer->neuronConnections,
                gpuPointer->neuronSections,
                gpuPointer->synapseConnections,
                gpuPointer->synapseSections,
                gpuPointer->updatePosSections,
                gpuPointer->segmentSettings,
                gpuPointer->inputTransfers,
                gpuPointer->outputTransfers,
                gpuPointer->randomValues,
                brickId);
        }
    }

    prcessOutputKernel<<<numberOfNeuronSections, 64>>>(
        gpuPointer->bricks,
        gpuPointer->neuronConnections,
        gpuPointer->neuronSections,
        gpuPointer->synapseConnections,
        gpuPointer->synapseSections,
        gpuPointer->updatePosSections,
        gpuPointer->segmentSettings,
        gpuPointer->outputTransfers,
        gpuPointer->randomValues);

    hipDeviceSynchronize();
    hipMemcpy(outputTransfers,
               gpuPointer->outputTransfers,
               segmentHeader->numberOfOutputTransfers * sizeof(float),
               hipMemcpyDeviceToHost);
}

extern "C"
void
backpropagation_CUDA(PointerHandler* gpuPointer,
                     SegmentSizes* segmentHeader,
                     uint32_t* brickOrder,
                     BrickHeader* bricks,
                     float* inputTransfers,
                     float* outputTransfers,
                     UpdatePosSection* updatePosSections,
                     const uint32_t numberOfNeuronSections)
{
    hipMemcpy(gpuPointer->inputTransfers,
               inputTransfers,
               segmentHeader->numberOfInputTransfers * sizeof(float),
               hipMemcpyHostToDevice);

    reweightOutputKernel<<<numberOfNeuronSections, NEURONS_PER_NEURONSECTION>>> (
        gpuPointer->bricks,
        gpuPointer->neuronSections,
        gpuPointer->inputTransfers);

    for(int32_t pos = segmentHeader->numberOfBricks - 1; pos >= 0; pos--)
    {
        const uint32_t brickId = brickOrder[pos];
        reweightCoreSegmentKernel<<<numberOfNeuronSections, 64>>>(
            gpuPointer->bricks,
            gpuPointer->neuronSections,
            gpuPointer->synapseConnections,
            gpuPointer->synapseSections,
            gpuPointer->segmentSettings,
            gpuPointer->inputTransfers,
            gpuPointer->outputTransfers,
            brickId);
    }

    hipDeviceSynchronize();
    hipMemcpy(outputTransfers,
               gpuPointer->outputTransfers,
               segmentHeader->numberOfOutputTransfers * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(updatePosSections,
               gpuPointer->updatePosSections,
               segmentHeader->numberOfUpdatePosSections * sizeof(UpdatePosSection),
               hipMemcpyDeviceToHost);
}

extern "C"
void
update_CUDA(PointerHandler* gpuPointer,
            SegmentSizes* segmentHeader,
            UpdatePosSection* updatePosSections,
            NeuronSection* neuronSections,
            SynapseConnection* synapseConnections,
            NeuronConnection* neuronConnections)
{
    hipMemcpy(gpuPointer->updatePosSections,
               updatePosSections,
               segmentHeader->numberOfUpdatePosSections * sizeof(UpdatePosSection),
               hipMemcpyHostToDevice);

    hipMemcpy(gpuPointer->neuronSections,
               neuronSections,
               segmentHeader->numberOfNeuronSections * sizeof(NeuronSection),
               hipMemcpyHostToDevice);

    hipMemcpy(gpuPointer->synapseConnections,
               synapseConnections,
               segmentHeader->numberOfSynapseSections * sizeof(SynapseConnection),
               hipMemcpyHostToDevice);

    hipMemcpy(gpuPointer->neuronConnections,
               neuronConnections,
               segmentHeader->numberOfNeuronSections * sizeof(NeuronConnection),
               hipMemcpyHostToDevice);
}
