
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define UUID_STR_LEN	37

// const predefined values
#define UNINIT_STATE_64 0xFFFFFFFFFFFFFFFF
#define UNINIT_STATE_32 0xFFFFFFFF
#define UNINIT_STATE_24 0xFFFFFF
#define UNINIT_STATE_16 0xFFFF
#define UNINIT_STATE_8  0xFF
#define UNINTI_POINT_32 0x0FFFFFFF

// common information
#define SYNAPSES_PER_SYNAPSESECTION 30
#define NEURONS_PER_NEURONSECTION 63
#define NEURON_CONNECTIONS 512
#define NUMBER_OF_RAND_VALUES 10485760
#define RAND_MAX 2147483647

enum SegmentTypes
{
    UNDEFINED_SEGMENT = 0,
    INPUT_SEGMENT = 1,
    OUTPUT_SEGMENT = 2,
    CORE_SEGMENT = 3,
};

enum ObjectTypes
{
    CLUSTER_OBJECT = 0,
    SEGMENT_OBJECT = 1,
};

struct SegmentHeaderEntry
{
    uint64_t bytePos = 0;
    uint64_t count = 0;

    // total size: 16 Byte
};

struct kuuid
{
    char uuid[UUID_STR_LEN];
    uint8_t padding[3];

    // total size: 40 Bytes
};

struct Position
{
    uint32_t x = UNINTI_POINT_32;
    uint32_t y = UNINTI_POINT_32;
    uint32_t z = UNINTI_POINT_32;
    uint32_t w = UNINTI_POINT_32;
};

struct SegmentHeader
{
    uint8_t objectType = SEGMENT_OBJECT;
    uint8_t version = 1;
    uint8_t segmentType = UNDEFINED_SEGMENT;
    uint8_t padding;
    uint32_t segmentID = UNINIT_STATE_32;
    uint64_t staticDataSize = 0;
    Position position;

    kuuid parentClusterId;

    // synapse-segment
    SegmentHeaderEntry name;
    SegmentHeaderEntry settings;
    SegmentHeaderEntry slotList;
    SegmentHeaderEntry inputTransfers;
    SegmentHeaderEntry outputTransfers;

    SegmentHeaderEntry bricks;
    SegmentHeaderEntry brickOrder;
    SegmentHeaderEntry neuronSections;
    SegmentHeaderEntry inputs;
    SegmentHeaderEntry outputs;
    SegmentHeaderEntry updatePosSections;

    SegmentHeaderEntry synapseSections;

    uint8_t padding2[246];

    // total size: 512 Byte
};

//==================================================================================================

typedef struct BrickHeader_struct
{
    // common
    uint brickId;
    bool isOutputBrick;
    bool isInputBrick;
    uint8_t padding1[14];
    uint neuronSectionPos;
    uint numberOfNeurons;
    uint numberOfNeuronSections;

    // total size: 32 Bytes
} BrickHeader;

//==================================================================================================

typedef struct Neuron_struct
{
    float input;
    float border;
    float potential;
    float delta;

    uint8_t refractionTime;
    uint8_t active;
    uint8_t padding[6];

    uint targetBorderId;
    uint targetSectionId;

    // total size: 32 Byte
} Neuron;

//==================================================================================================

typedef struct NeuronSection_struct
{
    Neuron neurons[NEURONS_PER_NEURONSECTION];
    uint numberOfNeurons;
    uint brickId;
    uint backwardNextId;
    uint8_t padding[20];

    // total size: 2048 Byte
} NeuronSection;

//==================================================================================================

typedef struct Synapse_struct
{
    float weight;
    float border;
    ushort targetNeuronId;
    char activeCounter;
    uint8_t padding[5];
    // total size: 16 Byte
} Synapse;

//==================================================================================================

typedef struct SynapseConnection_struct
{
    uint8_t active;
    uint8_t padding[3];

    float offset;
    uint randomPos;

    uint forwardNextId;
    uint backwardNextId;

    uint targetNeuronSectionId;
    uint sourceNeuronSectionId;
    uint sourceNeuronId;

    // total size: 32 Byte
} SynapseConnection;

//==================================================================================================

typedef struct SynapseSection_struct
{
    SynapseConnection connection;

    Synapse synapses[SYNAPSES_PER_SYNAPSESECTION];
    // total size: 512 Byte
} SynapseSection;

//==================================================================================================

typedef struct UpdatePos_struct
{
    uint type;
    uint randomPos;
    float offset;
    uint8_t padding[4];
    // total size: 16 Byte
} UpdatePos;

//==================================================================================================

typedef struct UpdatePosSection_struct
{
    UpdatePos positions[NEURONS_PER_NEURONSECTION];
    uint numberOfPositions;
    uint8_t padding[12];
    // total size: 1024 Byte
} UpdatePosSection;

//==================================================================================================

typedef struct SegmentSettings
{
    ulong maxSynapseSections;
    float synapseDeleteBorder;
    float neuronCooldown;
    float memorizing;
    float gliaValue;
    float signNeg;
    float potentialOverflow;
    float synapseSegmentation;
    float backpropagationBorder;
    uint8_t refractionTime;
    uint8_t doLearn;
    uint8_t updateSections;

    uint8_t padding[213];

    // total size: 256 Byte
} SegmentSettings;

//==================================================================================================

typedef struct NeuronSynapseConnection_struct
{
    uint backwardIds[NEURON_CONNECTIONS];
    // total size: 2048 Byte
} NeuronConnection;


//==================================================================================================
//==================================================================================================
//==================================================================================================

__device__ __forceinline__ int
getBlockId()
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    return index / blockDim.x;
}

/**
 * @brief initialize a new specific synapse
 */
__device__ __forceinline__ void
createNewSynapse(SynapseConnection* connection,
                 Synapse* synapse,
                 const NeuronSection* targetNeuronSection,
                 const SegmentSettings* segmentSettings,
                 const float outH,
                 const uint* randomValues)
{
    const float maxWeight = outH / (float)(segmentSettings->synapseSegmentation);

    // set activation-border
    connection->randomPos = (connection->randomPos + 1) % NUMBER_OF_RAND_VALUES;
    synapse->border = maxWeight * ((float)(randomValues[connection->randomPos]) / (float)(RAND_MAX));

    // set target neuron
    connection->randomPos = (connection->randomPos + 1) % NUMBER_OF_RAND_VALUES;
    synapse->targetNeuronId = (ushort)(randomValues[connection->randomPos]
                              % targetNeuronSection->numberOfNeurons);


    connection->randomPos = (connection->randomPos + 1) % NUMBER_OF_RAND_VALUES;
    synapse->weight = ((float)(randomValues[connection->randomPos]) / (float)(RAND_MAX)) / 10.0f;

    // update weight with sign
    connection->randomPos = (connection->randomPos + 1) % NUMBER_OF_RAND_VALUES;
    const uint signRand = randomValues[connection->randomPos] % 1000;
    synapse->weight *= (float)(1.0f - (1000.0f * segmentSettings->signNeg > signRand) * 2);

    synapse->activeCounter = 1;
}

//==================================================================================================

/**
 * @brief process synapse-section
 */
__device__ __forceinline__ void
synapseProcessingBackward(SynapseSection* section,
                          SynapseConnection* connection,
                          NeuronSection* targetNeuronSection,
                          NeuronSection* neuronSections,
                          UpdatePosSection* updatePosSections,
                          SegmentSettings* segmentSettings,
                          const uint* randomValues,
                          float* localMem)
{
    NeuronSection* sourceNeuronSection = &neuronSections[connection->sourceNeuronSectionId];
    Neuron* sourceNeuron = &sourceNeuronSection->neurons[connection->sourceNeuronId];
    const float sourcePotential = sourceNeuron->potential;

    float counter = connection->offset;
    uint pos = 0;

    // iterate over all synapses in the section
    while(pos < SYNAPSES_PER_SYNAPSESECTION
          && sourcePotential > counter)
    {
        Synapse* synapse = &section->synapses[pos];

        // create new synapse if necesarry and learning is active
        if(synapse->targetNeuronId == UNINIT_STATE_16)
        {
            createNewSynapse(connection,
                             synapse,
                             targetNeuronSection,
                             segmentSettings,
                             sourcePotential,
                             randomValues);
        }

        // update target-neuron
        Neuron* targetNeuron = &targetNeuronSection->neurons[synapse->targetNeuronId];
        //targetNeuron->input += synapse->weight;
        localMem[synapse->targetNeuronId] += synapse->weight;

        // update active-counter
        const uint8_t active = (synapse->weight > 0) == (targetNeuron->potential > targetNeuron->border);
        synapse->activeCounter += active * (uint8_t)(synapse->activeCounter < 126);

        // update loop-counter
        counter += synapse->border;
        pos++;
    }

    UpdatePosSection* updateSection = &updatePosSections[connection->sourceNeuronSectionId];
    UpdatePos* updatePos = &updateSection->positions[connection->sourceNeuronId];
    updatePos->type = sourcePotential - counter > 0.01f && connection->forwardNextId == UNINIT_STATE_32;
    updatePos->offset = counter + connection->offset;
}

//==================================================================================================

__device__ __forceinline__ void
prcessNeuronConnection(const uint neuronSectionId,
                       NeuronSection* targetNeuronSection,
                       NeuronConnection* neuronConnections,
                       NeuronSection* neuronSections,
                       SynapseConnection* synapseConnections,
                       SynapseSection* synapseSections,
                       UpdatePosSection* updatePosSections,
                       SegmentSettings* segmentSettings,
                       const uint* randomValues,
                       float* localMem)
{
    // reset weight of neurons
    for(uint neuronId = threadIdx.x;
        neuronId < targetNeuronSection->numberOfNeurons;
        neuronId += blockDim.x)
    {
        targetNeuronSection->neurons[neuronId].input = 0.0f;
    }

    for(uint sectionPos = threadIdx.x;
        sectionPos < NEURON_CONNECTIONS;
        sectionPos += blockDim.x)
    {
        // process synapse-sections
        const uint offset = threadIdx.x * NEURONS_PER_NEURONSECTION;
        const uint sectionId = neuronConnections[neuronSectionId].backwardIds[sectionPos];
        if(sectionId != UNINIT_STATE_32)
        {
            synapseProcessingBackward(&synapseSections[sectionId],
                                      &synapseConnections[sectionId],
                                      targetNeuronSection,
                                      neuronSections,
                                      updatePosSections,
                                      segmentSettings,
                                      randomValues,
                                      &localMem[offset]);
        }

        __syncthreads();

        // apply values of the local-memory to the neurons
        for(uint neuronId = threadIdx.x;
            neuronId < targetNeuronSection->numberOfNeurons;
            neuronId += blockDim.x)
        {
            Neuron* neuron = &targetNeuronSection->neurons[neuronId];
            for(uint i = neuronId;
                i < NEURONS_PER_NEURONSECTION * blockDim.x;
                i += NEURONS_PER_NEURONSECTION)
            {
                neuron->input += localMem[i];
                localMem[i] = 0.0f;
            }
        }
    }

    __syncthreads();
}

//==================================================================================================

__device__ __forceinline__ void
resetLocalMemory(float* localMem, const int localSize)
{
    // reset local memory
    for(uint i = threadIdx.x;
        i < localSize;
        i += blockDim.x)
    {
        localMem[i] = 0.0f;
    }
}

//==================================================================================================

/**
 * @brief process all neurons within a segment
 */
__global__ void
prcessCoreSegmentKernel(BrickHeader* bricks,
                        NeuronConnection* neuronConnections,
                        NeuronSection* neuronSections,
                        SynapseConnection* synapseConnections,
                        SynapseSection* synapseSections,
                        UpdatePosSection* updatePosSections,
                        SegmentSettings* segmentSettings,
                        float* inputTransfers,
                        float* outputTransfers,
                        const uint* randomValues,
                        const ulong brickId)
{
    __shared__ float localMem[4096];
    resetLocalMemory(localMem, blockDim.x * NEURONS_PER_NEURONSECTION);

    BrickHeader* brick = &bricks[brickId];
    if(brick->isInputBrick == false
            && brick->isOutputBrick == false)
    {
        for(uint neuronSectionId = brick->neuronSectionPos + getBlockId();
            neuronSectionId < brick->numberOfNeuronSections + brick->neuronSectionPos;
            neuronSectionId += gridDim.x)
        {
            NeuronSection* neuronSection = &neuronSections[neuronSectionId];

            prcessNeuronConnection(neuronSectionId,
                                   neuronSection,
                                   neuronConnections,
                                   neuronSections,
                                   synapseConnections,
                                   synapseSections,
                                   updatePosSections,
                                   segmentSettings,
                                   randomValues,
                                   localMem);

            for(uint neuronId = threadIdx.x;
                neuronId < neuronSection->numberOfNeurons;
                neuronId += blockDim.x)
            {
                Neuron* neuron = &neuronSection->neurons[neuronId];

                neuron->potential /= segmentSettings->neuronCooldown;
                neuron->refractionTime = neuron->refractionTime >> 1;

                if(neuron->refractionTime == 0)
                {
                    neuron->potential = segmentSettings->potentialOverflow * neuron->input;
                    neuron->refractionTime = segmentSettings->refractionTime;
                }

                // update neuron
                neuron->potential -= neuron->border;
                neuron->active = neuron->potential > 0.0f;
                neuron->input = 0.0f;
                neuron->potential = log2(neuron->potential + 1.0f);

                // handle active-state
                const bool needUpdate = neuron->active != 0 && neuron->targetSectionId == UNINIT_STATE_32;
                UpdatePos* updatePos = &updatePosSections[neuronSectionId].positions[neuronId];
                updatePos->type = needUpdate;
                updatePos->offset = 0.0f;
            }
        }
    }
}

//==================================================================================================

__global__ void
prcessOutputKernel(BrickHeader* bricks,
                   NeuronConnection* neuronConnections,
                   NeuronSection* neuronSections,
                   SynapseConnection* synapseConnections,
                   SynapseSection* synapseSections,
                   UpdatePosSection* updatePosSections,
                   SegmentSettings* segmentSettings,
                   float* outputTransfers,
                   const uint* randomValues)
{
    __shared__ float localMem[4096];
    resetLocalMemory(localMem, blockDim.x * NEURONS_PER_NEURONSECTION);

    NeuronSection* neuronSection = &neuronSections[getBlockId()];
    BrickHeader* brick = &bricks[neuronSection->brickId];
    if(brick->isOutputBrick)
    {
        prcessNeuronConnection(getBlockId(),
                               neuronSection,
                               neuronConnections,
                               neuronSections,
                               synapseConnections,
                               synapseSections,
                               updatePosSections,
                               segmentSettings,
                               randomValues,
                               localMem);

        for(uint neuronId = threadIdx.x;
            neuronId < neuronSection->numberOfNeurons;
            neuronId += blockDim.x)
        {
            Neuron* neuron = &neuronSection->neurons[neuronId];

            neuron->potential = segmentSettings->potentialOverflow * neuron->input;
            outputTransfers[neuron->targetBorderId] = neuron->potential;
            neuron->input = 0.0f;
        }
    }
}

//==================================================================================================

__global__ void
prcessInputKernel(BrickHeader* bricks,
                  NeuronSection* neuronSections,
                  UpdatePosSection* updatePosSections,
                  float* inputTransfers)
{
    NeuronSection* neuronSection = &neuronSections[getBlockId()];
    BrickHeader* brick = &bricks[neuronSection->brickId];
    const int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    if(brick->isInputBrick
            && threadIdx.x < neuronSection->numberOfNeurons)
    {
        Neuron* neuron = &neuronSection->neurons[threadIdx.x];
        neuron->potential = inputTransfers[neuron->targetBorderId];
        neuron->active = neuron->potential > 0.0f;

        // handle active-state
        const bool needUpdate = neuron->active != 0 && neuron->targetSectionId == UNINIT_STATE_32;
        UpdatePos* updatePos = &updatePosSections[getBlockId()].positions[threadIdx.x];
        updatePos->type = needUpdate;
        updatePos->offset = 0.0f;
    }
}

//==================================================================================================
//==================================================================================================
//==================================================================================================

/**
 * @brief run backpropagation for a single synapse-section
 */
__device__ __forceinline__ uint
backpropagateSection(SynapseSection* section,
                     SynapseConnection* connection,
                     Neuron* sourceNeuron,
                     const float outH,
                     const BrickHeader* brick,
                     NeuronSection* neuronSections,
                     SynapseConnection* synapseConnections,
                     SynapseSection* synapseSections)
{
    NeuronSection* targetNeuronSection = &neuronSections[connection->targetNeuronSectionId];
    float learnValue = 0.2f;
    float counter = connection->offset;

    // iterate over all synapses in the section
    for(uint32_t pos = 0; pos < SYNAPSES_PER_SYNAPSESECTION; pos++)
    {
        // break look, if no more synapses to process
        Synapse* synapse = &section->synapses[pos];

        if(outH > counter)
        {
            // update weight
            learnValue = (float)(126 - synapse->activeCounter) * 0.0002f;
            learnValue += 0.05f;
            Neuron* targetNeuron = &targetNeuronSection->neurons[synapse->targetNeuronId];
            sourceNeuron->delta += targetNeuron->delta * synapse->weight;

            synapse->weight -= learnValue * targetNeuron->delta;
        }

        counter += synapse->border;
    }

    return connection->forwardNextId;
}

//==================================================================================================

/**
 * @brief correct weight of synapses within a segment
 */
__global__ void
reweightCoreSegmentKernel(BrickHeader* bricks,
                          NeuronSection* neuronSections,
                          SynapseConnection* synapseConnections,
                          SynapseSection* synapseSections,
                          SegmentSettings* segmentSettings,
                          float* inputTransfers,
                          float* outputTransfers,
                          const ulong brickId)
{
    BrickHeader* brick = &bricks[brickId];

    for(uint neuronSectionId = brick->neuronSectionPos + getBlockId();
        neuronSectionId < brick->numberOfNeuronSections + brick->neuronSectionPos;
        neuronSectionId += gridDim.x)
    {
        NeuronSection* neuronSection = &neuronSections[neuronSectionId];
        for(uint neuronId = threadIdx.x;
            neuronId < neuronSection->numberOfNeurons;
            neuronId += blockDim.x)
        {
            Neuron* sourceNeuron = &neuronSection->neurons[neuronId];
            if(sourceNeuron->targetSectionId != UNINIT_STATE_32)
            {
                sourceNeuron->delta = 0.0f;
                if(sourceNeuron->active)
                {
                    uint nextId = sourceNeuron->targetSectionId;
                    while(nextId != UNINIT_STATE_32)
                    {
                        nextId = backpropagateSection(&synapseSections[nextId],
                                                      &synapseConnections[nextId],
                                                      sourceNeuron,
                                                      sourceNeuron->potential,
                                                      brick,
                                                      neuronSections,
                                                      synapseConnections,
                                                      synapseSections);
                    }

                    sourceNeuron->delta *= 1.4427f * pow(0.5f, sourceNeuron->potential);
                }

                if(brick->isInputBrick) {
                    outputTransfers[sourceNeuron->targetBorderId] = sourceNeuron->delta;
                }
            }
        }
    }
}

//==================================================================================================

__global__ void
reweightOutputKernel(BrickHeader* bricks,
                     NeuronSection* neuronSections,
                     float* inputTransfers)
{
    NeuronSection* neuronSection = &neuronSections[getBlockId()];
    BrickHeader* brick = &bricks[neuronSection->brickId];
    if(brick->isOutputBrick
            && threadIdx.x < neuronSection->numberOfNeurons)
    {
        Neuron* neuron = &neuronSection->neurons[threadIdx.x];
        neuron->delta = inputTransfers[neuron->targetBorderId];
        inputTransfers[neuron->targetBorderId] = 0.0f;
    }
}

struct PointerHandler
{
    BrickHeader* bricks = nullptr;
    uint32_t* brickOrder = nullptr;
    NeuronSection* neuronSections = nullptr;
    SynapseSection* synapseSections = nullptr;
    SegmentSettings* segmentSettings = nullptr;
    float* inputTransfers = nullptr;
    float* outputTransfers = nullptr;
    UpdatePosSection* updatePosSections = nullptr;
    uint32_t* randomValues = nullptr;
    NeuronConnection* neuronConnections = nullptr;
    SynapseConnection* synapseConnections = nullptr;
};

extern "C"
void
copyToDevice_CUDA(PointerHandler* gpuPointer,
                  SegmentHeader* segmentHeader,
                  SegmentSettings* segmentSettings,
                  BrickHeader* brickHeaders,
                  uint32_t* brickOrder,
                  NeuronSection* neuronSections,
                  SynapseSection* synapseSections,
                  UpdatePosSection* updatePosSections,
                  SynapseConnection* synapseConnections,
                  NeuronConnection* neuronConnections,
                  float* inputTransfers,
                  float* outputTransfers,
                  uint32_t* randomValues)
{
    hipMalloc(&gpuPointer->bricks,             segmentHeader->bricks.count             * sizeof(BrickHeader));
    hipMalloc(&gpuPointer->brickOrder,         segmentHeader->brickOrder.count         * sizeof(uint32_t));
    hipMalloc(&gpuPointer->neuronSections,     segmentHeader->neuronSections.count     * sizeof(NeuronSection));
    hipMalloc(&gpuPointer->synapseSections,    segmentHeader->synapseSections.count    * sizeof(SynapseSection));
    hipMalloc(&gpuPointer->segmentSettings,    1                                       * sizeof(SegmentSettings));
    hipMalloc(&gpuPointer->inputTransfers,     segmentHeader->inputTransfers.count     * sizeof(float));
    hipMalloc(&gpuPointer->outputTransfers,    segmentHeader->outputTransfers.count    * sizeof(float));
    hipMalloc(&gpuPointer->updatePosSections,  segmentHeader->updatePosSections.count  * sizeof(UpdatePosSection));
    hipMalloc(&gpuPointer->randomValues,       NUMBER_OF_RAND_VALUES                   * sizeof(uint32_t));
    hipMalloc(&gpuPointer->neuronConnections,  segmentHeader->neuronSections.count     * sizeof(NeuronConnection));
    hipMalloc(&gpuPointer->synapseConnections, segmentHeader->synapseSections.count    * sizeof(SynapseConnection));

    hipMemcpy(gpuPointer->bricks,             brickHeaders,       segmentHeader->bricks.count            * sizeof(BrickHeader),       hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->brickOrder,         brickOrder,         segmentHeader->brickOrder.count        * sizeof(uint32_t),          hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->neuronSections,     neuronSections,     segmentHeader->neuronSections.count    * sizeof(NeuronSection),     hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->synapseSections,    synapseSections,    segmentHeader->synapseSections.count   * sizeof(SynapseSection),    hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->segmentSettings,    segmentSettings,    1                                      * sizeof(SegmentSettings),   hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->inputTransfers,     inputTransfers,     segmentHeader->inputTransfers.count    * sizeof(float),             hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->outputTransfers,    outputTransfers,    segmentHeader->outputTransfers.count   * sizeof(float),             hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->updatePosSections,  updatePosSections,  segmentHeader->updatePosSections.count * sizeof(UpdatePosSection),  hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->randomValues,       randomValues,       NUMBER_OF_RAND_VALUES                  * sizeof(uint32_t),          hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->neuronConnections,  neuronConnections,  segmentHeader->neuronSections.count    * sizeof(NeuronConnection),  hipMemcpyHostToDevice);
    hipMemcpy(gpuPointer->synapseConnections, synapseConnections, segmentHeader->synapseSections.count   * sizeof(SynapseConnection), hipMemcpyHostToDevice);
}


extern "C"
void
processing_CUDA(PointerHandler* gpuPointer,
                SegmentHeader* segmentHeader,
                uint32_t* brickOrder,
                BrickHeader* bricks,
                float* inputTransfers,
                float* outputTransfers,
                const uint32_t numberOfNeuronSections)
{
    hipMemcpy(gpuPointer->inputTransfers,
               inputTransfers,
               segmentHeader->inputTransfers.count * sizeof(float),
               hipMemcpyHostToDevice);

    prcessInputKernel<<<numberOfNeuronSections, NEURONS_PER_NEURONSECTION>>>(
        gpuPointer->bricks,
        gpuPointer->neuronSections,
        gpuPointer->updatePosSections,
        gpuPointer->inputTransfers);

    const uint32_t numberOfBricks = segmentHeader->bricks.count;
    for(uint32_t pos = 0; pos < numberOfBricks; pos++)
    {
        const uint32_t brickId = brickOrder[pos];
        BrickHeader* brick = &bricks[brickId];
        if(brick->isInputBrick == false
                && brick->isOutputBrick == false)
        {
            prcessCoreSegmentKernel<<<10, 64>>>(
                gpuPointer->bricks,
                gpuPointer->neuronConnections,
                gpuPointer->neuronSections,
                gpuPointer->synapseConnections,
                gpuPointer->synapseSections,
                gpuPointer->updatePosSections,
                gpuPointer->segmentSettings,
                gpuPointer->inputTransfers,
                gpuPointer->outputTransfers,
                gpuPointer->randomValues,
                brickId);
        }
    }

    prcessOutputKernel<<<numberOfNeuronSections, 64>>>(
        gpuPointer->bricks,
        gpuPointer->neuronConnections,
        gpuPointer->neuronSections,
        gpuPointer->synapseConnections,
        gpuPointer->synapseSections,
        gpuPointer->updatePosSections,
        gpuPointer->segmentSettings,
        gpuPointer->outputTransfers,
        gpuPointer->randomValues);

    hipDeviceSynchronize();
    hipMemcpy(outputTransfers,
               gpuPointer->outputTransfers,
               segmentHeader->outputTransfers.count * sizeof(float),
               hipMemcpyDeviceToHost);
}

extern "C"
void
backpropagation_CUDA(PointerHandler* gpuPointer,
                     SegmentHeader* segmentHeader,
                     uint32_t* brickOrder,
                     BrickHeader* bricks,
                     float* inputTransfers,
                     float* outputTransfers,
                     UpdatePosSection* updatePosSections,
                     const uint32_t numberOfNeuronSections)
{
    hipMemcpy(gpuPointer->inputTransfers,
               inputTransfers,
               segmentHeader->inputTransfers.count * sizeof(float),
               hipMemcpyHostToDevice);

    reweightOutputKernel<<<numberOfNeuronSections, NEURONS_PER_NEURONSECTION>>> (
        gpuPointer->bricks,
        gpuPointer->neuronSections,
        gpuPointer->inputTransfers);

    const uint32_t numberOfBricks = segmentHeader->bricks.count;
    for(int32_t pos = numberOfBricks - 1; pos >= 0; pos--)
    {
        const uint32_t brickId = brickOrder[pos];
        reweightCoreSegmentKernel<<<numberOfNeuronSections, 64>>>(
            gpuPointer->bricks,
            gpuPointer->neuronSections,
            gpuPointer->synapseConnections,
            gpuPointer->synapseSections,
            gpuPointer->segmentSettings,
            gpuPointer->inputTransfers,
            gpuPointer->outputTransfers,
            brickId);
    }

    hipDeviceSynchronize();
    hipMemcpy(outputTransfers,
               gpuPointer->outputTransfers,
               segmentHeader->outputTransfers.count * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(updatePosSections,
               gpuPointer->updatePosSections,
               segmentHeader->updatePosSections.count * sizeof(UpdatePosSection),
               hipMemcpyDeviceToHost);
}

extern "C"
void
update_CUDA(PointerHandler* gpuPointer,
            SegmentHeader* segmentHeader,
            UpdatePosSection* updatePosSections,
            NeuronSection* neuronSections,
            SynapseConnection* synapseConnections,
            NeuronConnection* neuronConnections)
{
    hipMemcpy(gpuPointer->updatePosSections,
               updatePosSections,
               segmentHeader->updatePosSections.count * sizeof(UpdatePosSection),
               hipMemcpyHostToDevice);

    hipMemcpy(gpuPointer->neuronSections,
               neuronSections,
               segmentHeader->neuronSections.count * sizeof(NeuronSection),
               hipMemcpyHostToDevice);

    hipMemcpy(gpuPointer->synapseConnections,
               synapseConnections,
               segmentHeader->synapseSections.count * sizeof(SynapseConnection),
               hipMemcpyHostToDevice);

    hipMemcpy(gpuPointer->neuronConnections,
               neuronConnections,
               segmentHeader->neuronSections.count * sizeof(NeuronConnection),
               hipMemcpyHostToDevice);
}
